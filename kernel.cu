#include "hip/hip_runtime.h"
﻿/********************************************************************************/
// https://github.com/Allespro/ed25519_cuda
// test https://cyphr.me/ed25519_tool/ed.html
// PVK TEST: 3DBDF9B1912C999F1CD8289EDE88AAF4B930CD7EB41949A077F9F39B13935B05
// PUB TEST: F908FE4E03875446D863E059B7E95A087F8B3BE4FC9CACFBA9D05AE6B7A6F9FC
/********************************************************************************/



#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdint.h>
#include <stdio.h>
#include "ge.cuh"
#include "sc.cuh"
#include "kernel.cuh"

//unsigned int PERF_BLOCKS = 1024;
//unsigned int PERF_THREADS = 256;

//include "ed25519.cuh"

//include "onion.cuh"


__global__ void ed25519_test(int i) {
	i = 0;
	ge_p3 A;
	//***pva ed25519 test
	unsigned char public_key[32];
	//unsigned char private_key[64] = { 17, 17, 17, 17, 17, 17, 17, 17, 17, 17, 17, 17, 17, 17, 17, 17, 17, 17, 17, 17, 17, 17, 17, 17, 17, 17, 17, 17, 17, 17, 17, 17 };
	unsigned char private_key[64] = { 0x3d, 0xbd, 0xf9, 0xb1, 0x91, 0x2c, 0x99, 0x9f, 0x1c, 0xd8, 0x28, 0x9e, 0xde, 0x88, 0xaa, 0xf4, 0xb9, 0x30, 0xcd, 0x7e, 0xb4, 0x19, 0x49, 0xa0, 0x77, 0xf9, 0xf3, 
		0x9b, 0x13, 0x93, 0x5b, 0x05 };
	//unsigned char private_key[64] = "01234567890123456789012345678901";
	//unsigned char private_key[64] = {0};
	//printf("\n ED25519 TEST\n");
	printf("PVK TEST: 3dbdf9b1912c999f1cd8289ede88aaf4b930cd7eb41949a077f9f39b13935b05\n");

	printf("PVK ORIG: %02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X\n",
		private_key[0], private_key[1], private_key[2], private_key[3], private_key[4], private_key[5], private_key[6], private_key[7],
		private_key[8], private_key[9], private_key[10], private_key[11], private_key[12], private_key[13], private_key[14], private_key[15],
		private_key[16], private_key[17], private_key[18], private_key[19], private_key[20], private_key[21], private_key[22], private_key[23],
		private_key[24], private_key[25], private_key[26], private_key[27], private_key[28], private_key[29], private_key[30], private_key[31]);

	private_key[0] &= 248;
	private_key[31] &= 63;
	private_key[31] |= 64;

	printf("PVK Clamping: %02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X\n",
		private_key[0], private_key[1], private_key[2], private_key[3], private_key[4], private_key[5], private_key[6], private_key[7],
		private_key[8], private_key[9], private_key[10], private_key[11], private_key[12], private_key[13], private_key[14], private_key[15],
		private_key[16], private_key[17], private_key[18], private_key[19], private_key[20], private_key[21], private_key[22], private_key[23],
		private_key[24], private_key[25], private_key[26], private_key[27], private_key[28], private_key[29], private_key[30], private_key[31]);

	ge_scalarmult_base(&A, private_key);
	ge_p3_tobytes(public_key, &A);

	printf("PUB ED: %02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X%02X\n",
		public_key[0], public_key[1], public_key[2], public_key[3], public_key[4], public_key[5], public_key[6], public_key[7],
		public_key[8], public_key[9], public_key[10], public_key[11], public_key[12], public_key[13], public_key[14], public_key[15],
		public_key[16], public_key[17], public_key[18], public_key[19], public_key[20], public_key[21], public_key[22], public_key[23],
		public_key[24], public_key[25], public_key[26], public_key[27], public_key[28], public_key[29], public_key[30], public_key[31]);

	//printf("Correct D04AB232742BB4AB3A1368BD4615E4E6D0224AB71A016BAF8520A332C9778737");
	printf("Correct F908FE4E03875446D863E059B7E95A087F8B3BE4FC9CACFBA9D05AE6B7A6F9FC\n");
}
//*** ed25519 TEST

int main(int argc, char **argv) {

	ed25519_test << <1, 1>> > (1);
    	
	return 0;
}
